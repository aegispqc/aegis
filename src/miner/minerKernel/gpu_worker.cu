#include "hip/hip_runtime.h"
#include <iostream>
#include <bitset>
#include "./include/common.h"

#define UINT64 unsigned long long
#define UINT32 unsigned int
#define UINT8 unsigned char

#define GET_DEVICE_COUNT 0
#define GET_NUM_OF_EXECUTION 1
#define GET_X 2
#define GET_INFO 3

__device__ UINT64 unity = 1; 
__device__ UINT64 unityL;
__device__ int unuseZiro;


/**
 * Get the number of devices.
 */
int getDeviceCount()
{
    int deviceCount = 0;
    hipGetDeviceCount(&deviceCount);
    printf("Device count:%d", deviceCount);
    return deviceCount;
}

/**
 * Get the number of equations.
 */
UINT64 getNumOfExecution(int n, int num_block, int num_Thread)
{
    UINT64 unity = 1;
    UINT64 num_sm = num_block * num_Thread;
    UINT64 tableCount = (UINT64)((((unity << n) - 1) / num_sm) + 1);
    printf("Total number of executions:%llu", tableCount);
    return tableCount;
}

/**
 * Generation coefficient.
 */
void generateCoef(int n, int m, UINT64 *c, UINT64 *c0, char **EquCharArray)
{
    for (int i = 0; i < m; ++i)
    {
        char *EquChar = EquCharArray[i];
        UINT64 *EQU = (UINT64 *)malloc((n * (n + 1) / 2) * sizeof(UINT64));

        for (int j = 0; j < n * (n + 1) / 2; ++j)
        {
            EQU[j] = EquChar[j] - 48;
        }

        c0[i] = EquChar[(n * (n + 1) / 2)] - 48;
        int index = 0;
        for (int j = 0; j < n; ++j)
        {
            UINT64 temp2 = 0;
            for (int k = j; k < n; ++k)
            {
                temp2 ^= (EQU[index++] << (n - 1 - k));
            }
            c[i * n + j] = temp2;
        }
    }
}

/**
 * Generate a table of even numbers from 0 - 255.
 */
void generateByteParity(UINT64 *byte_parity)
{
    for (int i = 0; i < 256; ++i)
    {
        unsigned char temp = i;
        temp ^= temp >> 4;
        temp ^= temp >> 2;
        temp ^= temp >> 1;
        byte_parity[i] = temp & 1;
    }
}

/**
 * Receive equation coefficients in hexadecimal code from external sources.
 * Stop receiving when the 'END' string is received.
 */
void getEquations(int m, char **equations, int coefficientBit)
{
    int numOfBytes = coefficientBit / 8 + (coefficientBit % 8 == 0 ? 0 : 1);
    numOfBytes *= 2;

    char **cinBuf = (char**)malloc(sizeof(char*) * m);
    for (int i = 0; i < m; i++)
    {

        cinBuf[i] = (char*)malloc(sizeof(char) * (numOfBytes + 1));
        std::cin >> cinBuf[i];
    }
    
    char endBuf[32];
    std::cin >> endBuf;
    if (strncmp(endBuf, "end", 3))
    {
        fprintf(stderr, "Equations amount wrong!");
        exit(EXIT_FAILURE);
    }

    for (int i = 0; i < m; i++)
    {
        char* bs = hex2bin(cinBuf[i]);
        equations[i] = bs;
        
        free(cinBuf[i]);
    }
    
    free(cinBuf);
}

/**
 * Generate cuda index tables.
 */
void generateWhichXTable(UINT64 num_sm, UINT64 *whichXTable, UINT64 tableCount, int whichXWidth, UINT64 smCount = 0)
{
    whichXTable[0] = smCount * whichXWidth;
    for (UINT64 i = 1; i < num_sm; i++)
    {
        whichXTable[i] = whichXTable[i - 1] + tableCount;
    }
}

/**
 * Initializing cuda devices.
 */
bool Init(hipDeviceProp_t *deviceProp, int dev = 0)
{
    // set up device
    CHECK(hipGetDeviceProperties(deviceProp, dev));
    CHECK(hipSetDevice(dev));
    return true;
}

/**
 * Start Parallelizing Operations.
 */
__global__ void parallel(int n,
                         int m,
                         UINT64 num_sm,
                         int whichXWidth,
                         UINT64 *whichXTable,
                         UINT64 *__restrict__ c0,
                         UINT64 *__restrict__ c,
                         bool *resultFlg,
                         UINT64 *result)
{
    UINT32 ix = blockIdx.x * blockDim.x + threadIdx.x;
    UINT64 x = whichXTable[ix];
    if (*resultFlg)
        return;
    for (int w = 0; w < whichXWidth; w++)
    {
        UINT64 *p;
        p = c;
        int i;
        UINT64 tempX = x << unuseZiro;
        for (i = 0; i < m; ++i)
        {
            UINT64 y = 0;
            UINT64 tmptmpX = tempX;
            while (tmptmpX)
            {
                int pos1 = __clzll(tmptmpX);
                tmptmpX ^= unityL >> pos1;
                y ^= (p[pos1] & x);
            }
            if ((__popcll(y) & 1) != c0[i])
                break;
            p += n;
        }
        if (i == m)
        {
            *resultFlg = true;
            *result = x;
            return;
        }
        x++;
    }
    whichXTable[ix] = x;
}

/**
 * Get x and print out json format.
 */
void getX(int m, int n, int num_block, int num_Thread, int whichXWidth, UINT64 startSMCount, int coefficientBit)
{
    UINT64 num_sm = num_block * num_Thread;

    UINT64 unity = 1;
    UINT64 unityLhost = unity << 63;
    int unuseZiroHost = 64 - n;
    UINT64 *c0, *c, *byte_parity, *result;
    char **equations;
    bool *resultFlg;
    UINT64 *whichXTable;
    UINT64 tableCount = (UINT64)((((unity << n) - 1) / num_sm) + 1);
    
    if (tableCount < whichXWidth)
    {
        whichXWidth = 1;
    }

    CHECK(hipMallocManaged(&c, m * n * sizeof(UINT64)));
    CHECK(hipMallocManaged(&c0, m * sizeof(UINT64)));
    CHECK(hipMallocManaged(&byte_parity, 256 * sizeof(UINT64)));
    CHECK(hipMallocManaged(&whichXTable, num_sm * sizeof(UINT64)));
    CHECK(hipMallocManaged(&result, sizeof(UINT64)));
    CHECK(hipMallocManaged(&resultFlg, sizeof(bool)));
    hipMemcpyToSymbol(HIP_SYMBOL(unityL), &unityLhost, sizeof(UINT64), 0, hipMemcpyHostToDevice);
    hipMemcpyToSymbol(HIP_SYMBOL(unuseZiro), &unuseZiroHost, sizeof(int), 0, hipMemcpyHostToDevice);

    *resultFlg = false;
    equations = (char **)malloc(m * sizeof(char *));

    UINT64 start, lastTime = 0, end = 0;
    start = dtime_msec(0);
    getEquations(m, equations, coefficientBit);
    generateCoef(n, m, c, c0, equations);
    generateWhichXTable(num_sm, whichXTable, tableCount, whichXWidth, startSMCount);
    generateByteParity(byte_parity);

    UINT64 solution = 0;
    UINT64 smCount = 0, maxSmCount = ceil(tableCount / (double)whichXWidth);
    start = dtime_msec(0);

    for (smCount = startSMCount; smCount < maxSmCount; smCount++)
    {
        parallel<<<num_block, num_Thread>>>(n, m, num_sm, whichXWidth, whichXTable, c0, c, resultFlg, result);
        CHECK(hipDeviceSynchronize());
        if (*resultFlg)
        {
            solution = *result;
            break;
        }
        lastTime = end;
        end = dtime_msec(start);
    }
    end = dtime_msec(start);

    if (*resultFlg)
    {
        double rate = 0.0;
        UINT64 smUse = ((smCount - startSMCount + 1) * whichXWidth * num_sm);
        if (smCount - startSMCount != 0 && lastTime != 0)
        {
            rate = ((smCount - startSMCount) * whichXWidth * num_sm) / ((float)lastTime / CLOCKS_PER_SEC);
        }

        std::bitset<64> solBitset(solution);
        printf("x found:{\"x\":\"%s\", \"gpuTime\":\"%f\", \"rate\":\"%f\", \"smCount\":\"%llu\", \"smUse\":\"%llu\"}\n",
               solBitset.to_string().c_str(),
               end / (float)CLOCKS_PER_SEC,
               rate,
               smCount,
               smUse);
		fflush(stdout);
    }
    else
    {
        printf("x not found\n");
    }
	
    hipFree(c);
    hipFree(c0);
    hipFree(byte_parity);
    hipFree(whichXTable);
    hipFree(result);
    hipFree(resultFlg);
    for (int i = 0; i < m; i++)
		free(equations[i]);
	free(equations);
}

int main(int argc, char **argv)
{
    int opt = 0, deviceID = 0, m = 1, n = 1, whichXWidth = 1;
    int coefficientBit = 0;
    UINT64 startSMCount = 0;
    for (int i = 1; i < argc; i++)
    {
        switch (i)
        {
        case 1:
            opt = atoi(argv[i]);
            break;
        case 2:
            deviceID = atoi(argv[i]);
            break;
        case 3:
            m = atoi(argv[i]);
            break;
        case 4:
            n = atoi(argv[i]);
            break;
        case 5:
            whichXWidth = atoi(argv[i]);
            break;
        case 6:
            startSMCount = strtoull(argv[i], NULL, 10);
            break;
        case 7:
            coefficientBit = atoi(argv[i]);
            break;
        default:
            break;
        }
    }

    if (argc < 2)
    {
        printf("Insufficient parameters!\n");
        return EXIT_FAILURE;
    }
    hipDeviceProp_t deviceProp;
    Init(&deviceProp, deviceID);

    int num_block = deviceProp.multiProcessorCount * 128;
    int num_Thread = 1024;

    switch (opt)
    {
    case GET_DEVICE_COUNT: 
        getDeviceCount();
        break;
    case GET_NUM_OF_EXECUTION: 
        getNumOfExecution(n, num_block, num_Thread);
        break;
    case GET_X:
        getX(m, n, num_block, num_Thread, whichXWidth, startSMCount, coefficientBit);
        break;
    default:
        printf("unknow opt!\n");
        return EXIT_FAILURE;
    }

    return EXIT_SUCCESS;
}